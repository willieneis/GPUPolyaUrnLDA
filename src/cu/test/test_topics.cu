#include "hip/hip_runtime.h"
#include "test_topics.cuh"
#include "../topics.cuh"
#include "../train.cuh"
#include "../random.cuh"
#include "../error.cuh"
#include "assert.h"

using gpulda::FileLine;
using gpulda::f32;
using gpulda::i32;
using gpulda::u32;
using gpulda::u64;

namespace gpulda_test {

void test_compute_d_idx() {
  u32 size = 4*GPULDA_COMPUTE_D_IDX_BLOCKDIM;
  u32 d_len[4*GPULDA_COMPUTE_D_IDX_BLOCKDIM];
  u32 d_idx[4*GPULDA_COMPUTE_D_IDX_BLOCKDIM];
  u32 n_docs = 2*GPULDA_COMPUTE_D_IDX_BLOCKDIM + 15;

  for(i32 i = 0; i < size; ++i) {
    d_len[i] = i+1;
  }

  u32* gpu_d_len;
  u32* gpu_d_idx;
  hipMalloc(&gpu_d_len, size*sizeof(u32)) >> GPULDA_CHECK;
  hipMalloc(&gpu_d_idx, size*sizeof(u32)) >> GPULDA_CHECK;

  hipMemcpy(gpu_d_len, d_len, size*sizeof(u32), hipMemcpyHostToDevice) >> GPULDA_CHECK;

  gpulda::compute_d_idx<<<1,GPULDA_COMPUTE_D_IDX_BLOCKDIM>>>(gpu_d_len, gpu_d_idx, n_docs);
  hipDeviceSynchronize() >> GPULDA_CHECK;

  hipMemcpy(d_idx, gpu_d_idx, size*sizeof(u32), hipMemcpyDeviceToHost) >> GPULDA_CHECK;

  assert(d_idx[0] == 0);
  u32 j = d_len[0];
  for(i32 i = 1; i < n_docs; ++i) {
     assert(d_idx[i] == j);
     j = j + d_len[i];
   }

  hipFree(gpu_d_len);
  hipFree(gpu_d_idx);
}

__global__ void test_draw_alias(u32* error) {
  // compute constants
  constexpr u32 size = 5;
  __shared__ f32 prob[size];
  __shared__ u32 alias[size];

  // build alias table
  for(i32 offset = 0; offset < size / blockDim.x + 1; ++offset) {
    i32 i = offset * blockDim.x + threadIdx.x;
    if(i<size) {
      prob[i] = 0.5;
      alias[i] = 1;
    }
  }
  __syncthreads();

  // draw from prob
  u32 topic = gpulda::draw_alias(0.6, prob, alias, size);
  if(threadIdx.x==0 && topic!=3){
    error[0] = 1;
  }

  // draw from alias
  topic = gpulda::draw_alias(0.75, prob, alias, size);
  if(threadIdx.x==0 && topic!=1){
    error[0] = 2;
  }
}

__global__ void test_draw_wary_search(u32* error) {
  constexpr i32 size = 96; // hashmap size must be multiple of 16
  __shared__ gpulda::HashMap m;
  __shared__ u64 data[size];
  __shared__ f32 mPhi[size];
  m.capacity = size;
  m.data = data;
  m.temp_data = mPhi;
  f32 sigma_b = 50.0f;

  for(i32 offset = 0; offset < size / blockDim.x + 1; ++offset) {
    i32 i = offset * blockDim.x + threadIdx.x;
    if(i<size) {
      data[i] = m.entry(i,0);
      mPhi[i] = ((float)i) * sigma_b / ((float)size);
    }
  }
  __syncthreads();

  // test standard case: first entry in first slot
  u32 topic = gpulda::draw_wary_search(0.0f, &m, sigma_b);
  if(threadIdx.x==0 && topic!=0){
    error[0] = 1;
  }

  // test standard case: second entry in first slot
  topic = gpulda::draw_wary_search(0.02f, &m, sigma_b);
  if(threadIdx.x==0 && topic!=1){
    error[0] = 2;
  }

  // test edge case 1: last entry in first slot, search ends in second slot
  topic = gpulda::draw_wary_search(0.16f, &m, sigma_b);
  if(threadIdx.x==0 && topic!=15){
    error[0] = 3;
  }

  // test standard case: value in middle of slot
  topic = gpulda::draw_wary_search(0.4f, &m, sigma_b);
  if(threadIdx.x==0 && topic!=38){
    error[0] = 4;
  }

  // test standard case: second-to-last entry in last slot
  topic = gpulda::draw_wary_search(0.985f, &m, sigma_b);
  if(threadIdx.x==0 && topic!=94){
    error[0] = 5;
  }

  // test edge case 2: last entry in last slot
  topic = gpulda::draw_wary_search(1.0f, &m, sigma_b);
  if(threadIdx.x==0 && topic!=95){
    error[0] = 6;
  }
}

__global__ void test_count_topics(u32* error) {
  // compute constants
  __shared__ hiprandStatePhilox4_32_10_t rng;
  if(threadIdx.x == 0) {
    hiprand_init((unsigned long long) 0, (unsigned long long) 0, (unsigned long long) 0, &rng);
  }
  __syncthreads();
  constexpr u32 cutoff = 25;
  __shared__ u32 count[cutoff];

  // declare arguments
  constexpr u32 size = 100;
  __shared__ u32 z[size];

  // initialize hashmap
  __shared__ gpulda::HashMap m;
  m.init(size, &rng);
  __syncthreads();

  // prepare state
  for(i32 offset = 0; offset < size / blockDim.x + 1; ++offset) {
    i32 i = offset * blockDim.x + threadIdx.x;
    if(i < size) {
      z[i] = i % cutoff;
    }
  }
  __syncthreads();

  // test count_topics
  gpulda::count_topics(z, size, &m);
  __syncthreads();

  // retrieve values
  for(i32 offset = 0; offset < blockDim.x + 1; ++offset) {
    i32 i = offset * blockDim.x + threadIdx.x;

    // retreive from hashmap, two half-lanes at a time
    for(i32 j = 0; j < warpSize/2; ++j) {
      i32 warp_i = __shfl(i, j, warpSize/2);
      u32 warp_ct = m.get2(warp_i);
      if(i == warp_i && i < cutoff) {
        count[i] = warp_ct;
      }
    }
  }

  // check correctness
  if(threadIdx.x == 0) {
    for(i32 i = 0; i < cutoff; ++i) {
      if(count[i] != size/cutoff) {
        error[0] = i+1;
        break;
      }
    }
  }
}

__global__ void test_compute_product_cumsum(u32* error) {
  // compute constants
  __shared__ f32 block_scan_temp[GPULDA_SAMPLE_TOPICS_BLOCKDIM / GPULDA_BLOCK_SCAN_WARP_SIZE];
  f32 tolerance = 0.0001f; // large to allow for randomness

  // declare arguments
  constexpr u32 size = 100;
  __shared__ f32 Phi_dense[size];
  __shared__ f32 mPhi[size];
  __shared__ f32 check[size];

  // populate hashmap data
  __shared__ gpulda::HashMap m;
  __shared__ u64 data[size];
  m.capacity = size;
  m.data = data;
  m.temp_data = mPhi;

  // prepare state
  for(i32 offset = 0; offset < size / blockDim.x + 1; ++offset) {
    i32 i = offset * blockDim.x + threadIdx.x;
    if(i < size) {
      Phi_dense[i] = 6.0f * (float) i;
      data[i] = m.entry(i, i);
      check[i] = (i == 0) ? 0.0f : ((float) (i-1)) * (((float) (i-1))+1.0f) * ((2.0f*((float) (i-1)))+1.0f);
    }
  }
  __syncthreads();

  // test count_topics
  f32 total = gpulda::compute_product_cumsum(&m, Phi_dense, block_scan_temp);
  __syncthreads();

  // check correctness
  if(threadIdx.x == 0) {
    for(i32 i = 0; i < size; ++i) {
      if(abs(mPhi[i] - check[i]) > tolerance) {
        error[0] = i+1;
        break;
      }
    }
    f32 expected_total = ((float) (size-1)) * (((float) (size-1))+1.0f) * ((2.0f*((float) (size-1)))+1.0f);
    if(total != expected_total) {
      error[0] = size+1;
    }
  }
}

void test_sample_topics_functions() {
  u32* out;
  hipMalloc(&out, sizeof(u32)) >> GPULDA_CHECK;
  u32 out_host = 0;

  // draw topic via Alias table
  test_draw_alias<<<1,GPULDA_SAMPLE_TOPICS_BLOCKDIM>>>(out);
  hipDeviceSynchronize() >> GPULDA_CHECK;

  hipMemcpy(&out_host, out, sizeof(u32), hipMemcpyDeviceToHost) >> GPULDA_CHECK;
  assert(out_host == 0);

  // draw topic via wary search
  test_draw_wary_search<<<1,GPULDA_SAMPLE_TOPICS_BLOCKDIM>>>(out);
  hipDeviceSynchronize() >> GPULDA_CHECK;

  hipMemcpy(&out_host, out, sizeof(u32), hipMemcpyDeviceToHost) >> GPULDA_CHECK;
  assert(out_host == 0);

  // count topics
  test_count_topics<<<1,GPULDA_SAMPLE_TOPICS_BLOCKDIM>>>(out);
  hipDeviceSynchronize() >> GPULDA_CHECK;

  hipMemcpy(&out_host, out, sizeof(u32), hipMemcpyDeviceToHost) >> GPULDA_CHECK;
  assert(out_host == 0);

  // compute sparse vector product
  test_compute_product_cumsum<<<1,GPULDA_SAMPLE_TOPICS_BLOCKDIM>>>(out);
  hipDeviceSynchronize() >> GPULDA_CHECK;

  hipMemcpy(&out_host, out, sizeof(u32), hipMemcpyDeviceToHost) >> GPULDA_CHECK;
  assert(out_host == 0);

  // cleanup
  hipFree(out);
}

void test_sample_topics() {
  constexpr f32 alpha = 0.1f;
  constexpr f32 beta = 0.1f;
  constexpr u32 V = 3;
  constexpr u32 K = 5;
  u32 C[V] = {1,1,1};
  constexpr u32 buffer_size = 5;
  constexpr u32 max_D = 2;

  gpulda::Args args = {alpha,beta,K,V,C,buffer_size,max_D};
  u32 z[buffer_size] = {4,1,0,4,0};
  u32 w[buffer_size] = {0,0,0,0,0};
  u32 d[max_D] = {3,2};
  u32 K_d[max_D] = {1,1};
  u32 n_docs = max_D;
  u32 n_tokens = buffer_size;
  gpulda::Buffer buffer = {z, w, d, K_d, n_docs, n_tokens, NULL, NULL, NULL, NULL, NULL, NULL, NULL};

  gpulda::initialize(&args, &buffer, 1);

  // initialize test-specific Phi
  f32 Phi_host[K*V] = { 0.98f, 0.02f, 0.02f, 0.02f, 0.02f,
                        0.01f, 0.49f, 0.49f, 0.49f, 0.49f,
                        0.01f, 0.49f, 0.49f, 0.49f, 0.49f };
  f32* Phi_dense;
  hipMalloc(&Phi_dense, K*V*sizeof(f32)) >> GPULDA_CHECK;
  hipMemcpy(Phi_dense, Phi_host, K*V*sizeof(f32), hipMemcpyHostToDevice) >> GPULDA_CHECK;
  u32* n_dense;
  hipMalloc(&n_dense, K*V*sizeof(u32)) >> GPULDA_CHECK;

  // initialize test-specific sigma_a
  f32 sigma_a_host[V] = { 0.0f, 0.0f, 0.0f };
  f32* sigma_a;
  hipMalloc(&sigma_a, V*sizeof(f32)) >> GPULDA_CHECK;
  hipMemcpy(sigma_a, sigma_a_host, V*sizeof(f32), hipMemcpyHostToDevice) >> GPULDA_CHECK;

  // copy z,w,d to buffer
  hipMemcpy(buffer.gpu_z, z, buffer_size*sizeof(u32), hipMemcpyHostToDevice) >> GPULDA_CHECK;
  hipMemcpy(buffer.gpu_w, w, buffer_size*sizeof(u32), hipMemcpyHostToDevice) >> GPULDA_CHECK;
  hipMemcpy(buffer.gpu_d_len, d, n_docs*sizeof(u32), hipMemcpyHostToDevice) >> GPULDA_CHECK;
  hipMemcpy(buffer.gpu_K_d, K_d, n_docs*sizeof(u32), hipMemcpyHostToDevice) >> GPULDA_CHECK;
  gpulda::compute_d_idx<<<1,GPULDA_COMPUTE_D_IDX_BLOCKDIM>>>(buffer.gpu_d_len, buffer.gpu_d_idx, n_docs);
  hipDeviceSynchronize() >> GPULDA_CHECK;

  // sample a topic indicator
  gpulda::sample_topics<<<n_docs,GPULDA_SAMPLE_TOPICS_BLOCKDIM>>>(args.buffer_size, buffer.gpu_z, buffer.gpu_w, buffer.gpu_d_len, buffer.gpu_d_idx, buffer.gpu_K_d, V, n_dense, Phi_dense, sigma_a, NULL, NULL, 0, buffer.gpu_rng);
  hipDeviceSynchronize() >> GPULDA_CHECK;

  hipMemcpy(z, buffer.gpu_z, buffer_size*sizeof(u32), hipMemcpyDeviceToHost) >> GPULDA_CHECK;
  for(i32 i = 0; i < buffer_size; ++i) {
    assert(z[i] == 0);
  }

  // cleanup
  hipFree(Phi_dense);
  hipFree(n_dense);
  hipFree(sigma_a);
  gpulda::cleanup(&buffer, 1);
}

}
